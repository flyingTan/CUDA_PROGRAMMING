#include "hip/hip_runtime.h"
#include "add.hpp"

__global__ void square(float* d_in, float* d_out) {
    int idx =  threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;
}

void square_run(float* d_in, float* d_out, const int d_buffsize){
    square<<<1, d_buffsize>>>(d_in, d_out);
}